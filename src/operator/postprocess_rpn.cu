#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file postprocess_rpn.cu
 * \brief post process of rpn operator
 * \author Ming Zhang
*/
#include "./postprocess_rpn-inl.h"
#include "./mshadow_op.h"


namespace mshadow {

namespace cuda {

__global__ void PostProcessRPNForwardKernel1() {
}


__global__ void PostProcessRPNForwardKernel(
                int count,
                const float *pfCls, const float *pfReg, 
                const float *pfAnchor, const float *pfOtherinfo, 
                int dwAnchorNum, int dwFeatH, int dwFeatW, 
                float *pfBBs, int dwMaxBBNum, int *pdwbb_num_now) {
#if 1     
  float clsthreshold = pfOtherinfo[0];
  int originalH = pfOtherinfo[1];
  int originalW = pfOtherinfo[2]; 
//  printf("clsthreshold:%.1f, originalH:%d, originalW:%d\n", clsthreshold, originalH, originalW);
//  __syncthreads();
  int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

  if (1 && index < count)
  {
    int nownum = *pdwbb_num_now;
//    printf("bidxx:%d-bidxy:%d-gdimx:%d-bdimx:%d-tidxx:%d, index:%d, nownum:%d\n", blockIdx.x, blockIdx.y, gridDim.x, blockDim.x, threadIdx.x, index, nownum);
//    printf("bidxx:%d-bidxy:%d-gdimx:%d-bdimx:%d-tidxx:%d, index:%d\n", blockIdx.x, blockIdx.y, gridDim.x, blockDim.x, threadIdx.x, index);
//    __syncthreads();
#if 1
    if (nownum >= 0 && nownum < dwMaxBBNum)
    {
      int dwFeatSize = dwFeatH * dwFeatW;
      int dwAnchorI = index / dwFeatSize;
      int dwRI = (index - dwAnchorI * dwFeatSize) / dwFeatW;
      int dwCI = (index - dwAnchorI * dwFeatSize) % dwFeatW;
      int dwOft = dwRI * dwFeatW + dwCI;
      int dwAnchorOft = dwAnchorI * dwFeatSize;
      const float *pfNowAnchor = pfAnchor + dwAnchorI * 2;
      if (pfCls[dwOft + dwAnchorOft] > clsthreshold)
      {
        float fCY = pfReg[dwAnchorOft * 4 + 0 * dwFeatSize + dwOft];
        float fCX = pfReg[dwAnchorOft * 4 + 1 * dwFeatSize + dwOft];
        float fH = pfReg[dwAnchorOft * 4 + 2 * dwFeatSize + dwOft];
        float fW = pfReg[dwAnchorOft * 4 + 3 * dwFeatSize + dwOft];
        fCY = fCY * pfNowAnchor[0] + ((float)(dwRI) * originalH) / dwFeatH;
        fCX = fCX * pfNowAnchor[1] + ((float)(dwCI) * originalW) / dwFeatW;
        fH = expf(fH) * pfNowAnchor[0];
        fW = expf(fW) * pfNowAnchor[1];
        atomicInc((unsigned int*)pdwbb_num_now, dwMaxBBNum);
        nownum = *pdwbb_num_now;
       
        if (nownum > 0)
        {
          pfBBs[(nownum-1) * 4 + 0] = fCY;
          pfBBs[(nownum-1) * 4 + 1] = fCX;
          pfBBs[(nownum-1) * 4 + 2] = fH;
          pfBBs[(nownum-1) * 4 + 3] = fW;

//          printf("bidxx:%d-bidxy:%d-gdimx:%d-bdimx:%d-tidxx:%d, index:%d, nownum:%d\n", blockIdx.x, blockIdx.y, gridDim.x, blockDim.x, threadIdx.x, index, nownum);
//          __syncthreads();
        }
      }
    }
#endif
  }
#endif
}


inline void PostProcessRPNForward(const Tensor<gpu, 4> &datacls_in,
                           const Tensor<gpu, 4> &datareg_in,
                           const Tensor<gpu, 2> &anchorinfo_in,
                           const Tensor<gpu, 1> &otherinfo_in,
                           Tensor<gpu, 3> &bb_out) {
  CHECK_EQ(datacls_in.size(0), datareg_in.size(0));

  int dwBatchNum = datacls_in.size(0);
  int dwAnchorNum = anchorinfo_in.size(0);
  int bb_maxnum_per_batch = bb_out.size(1);
  
  int dwFeatH = datacls_in.size(2);
  int dwFeatW = datacls_in.size(3);
  int dwBBMemLen = bb_out.MSize();
  hipMemset(bb_out.dptr_, 0, dwBBMemLen*sizeof(float));
  int *pdwCounter = 0;
  hipMalloc(&pdwCounter, dwBatchNum*sizeof(int));
  hipMemset(pdwCounter, 0, dwBatchNum*sizeof(int));
//  printf("dwBBMemLen:%d\n", dwBBMemLen);
//  float *pfAnchorData = anchorinfo_in.dptr_;
//  printf("anchor[%dx%d]_address:%x\n", anchorinfo_in.size(0), anchorinfo_in.size(1), pfAnchorData);
//  for (int dwI = 0; dwI < 1; dwI++)
//  {
//    printf("anchor_%d:%.1f, %.1f\n", dwI, pfAnchorData[dwI * 2 + 0], pfAnchorData[dwI * 2 + 1]);
//  }
//  __syncthreads();
  
  int count = dwFeatH * dwFeatW * dwAnchorNum;
#if 1
  const int gridSize = (count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  dim3 dimGrid(kMaxGridNum, (gridSize + kMaxGridNum - 1) / kMaxGridNum);
  dim3 dimBlock(kMaxThreadsPerBlock);
#else
  dim3 dimGrid(2, 2);
  dim3 dimBlock(2);
#endif
  CheckLaunchParam(dimGrid, dimBlock, "PostProcessRPN Forward");
  hipStream_t stream = Stream<gpu>::GetStream(bb_out.stream_);
  
  for (int bi = 0; bi < dwBatchNum; bi++) {
//    printf("fucking start...(%d)\n", bi);
    const Tensor<gpu, 3> &datacls_onebatch = datacls_in[bi];
    const Tensor<gpu, 3> &datareg_onebatch = datareg_in[bi];
    Tensor<gpu, 2> bb_onebatch = bb_out[bi];
//    printf("count:%d\n", count);
//    printf("datacls_onebatch.dptr_:%x\n", datacls_onebatch.dptr_);
//    printf("datareg_onebatch.dptr_:%x\n", datareg_onebatch.dptr_);
//    printf("anchorinfo_in.dptr_:%x\n", anchorinfo_in.dptr_);
//    printf("otherinfo_in.dptr_:%x\n", otherinfo_in.dptr_);
//    printf("dwAnchorNum:%d, dwFeatH:%d, dwFeatW:%d\n", dwAnchorNum, dwFeatH, dwFeatW);
//    printf("bb_onebatch.dptr_:%x\n", bb_onebatch.dptr_);
//    printf("bb_maxnum_per_batch:%d\n", bb_maxnum_per_batch); 
#if 1   
    PostProcessRPNForwardKernel<<<dimGrid, dimBlock, 0, stream>>>(
            count, 
            datacls_onebatch.dptr_, datareg_onebatch.dptr_, 
            anchorinfo_in.dptr_, otherinfo_in.dptr_, dwAnchorNum, dwFeatH, dwFeatW, 
            bb_onebatch.dptr_, bb_maxnum_per_batch, pdwCounter+bi);
//    hipDeviceSynchronize();
#else
    PostProcessRPNForwardKernel1<<<dimGrid, dimBlock>>>();
#endif
//    printf("fucking ending...(%d)\n", bi);
  }
  hipFree(pdwCounter);
}
  
} // namespace cuda

inline void PostProcessRPNForward(const Tensor<gpu, 4> &datacls_in,
                           const Tensor<gpu, 4> &datareg_in,
                           const Tensor<gpu, 2> &anchorinfo_in,
                           const Tensor<gpu, 1> &otherinfo_in,
                           Tensor<gpu, 3> &bb_out) {
//  printf("originalW:%d\n", originalW);                           
  cuda::PostProcessRPNForward(datacls_in, datareg_in, anchorinfo_in, otherinfo_in, bb_out);
}

} // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator *CreateOp<gpu>(PostProcessRPNParam param) {
  return new PostProcessRPNOp<gpu>(param);
}

}  // namespace op
}  // namespace mxnet
